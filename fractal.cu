#include "hip/hip_runtime.h"
#include <GLUT/glut.h>
#include <math.h>
#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

struct complex {
	float real;
	float imag;	
};

const int width = 1600;

float *z = (float*) malloc(width * width * sizeof(float));

__host__ __device__ float abs(complex x) {
	return sqrt(x.real * x.real + x.imag * x.imag);
}

__host__ __device__ complex operator*(complex x, complex y) {
	complex c = {
	(x.real * y.real - x.imag * y.imag),
	(x.imag * y.real + x.real * y.imag)
	};
	return c;
}

__host__ __device__ complex operator*(complex x, float dy) {
	complex y = {dy, 0.0};
	return x*y;
}

__host__ __device__ complex operator/(complex x, complex y) {
	complex c = {
	(x.real * y.real + x.imag * y.imag) / (y.real * y.real + y.imag * y.imag),
	(x.imag * y.real - x.real * y.imag) / (y.real * y.real + y.imag * y.imag)
	};
	return c;
}

__host__ __device__ complex operator+(complex x, complex y) {
	complex c = {
	(x.real + y.real),
	(x.imag + y.imag)
	};
	return c;
}

__host__ __device__ complex operator-(complex x, complex y) {
	complex c = {
	(x.real - y.real),
	(x.imag - y.imag)
	};
	return c;
}

__host__ __device__ complex operator-(complex x, float dy) {
	complex y = {dy, 0.0};
	return x-y;
}

__host__ __device__ complex f(complex x) {
	complex d = x*x*x*x*x*x*x*x + x*x*x*x*x*x*15.0 - 16.0;
	return d;
}

__host__ __device__ complex df(complex x) {
  	complex d = x*x*x*x*x*x*x*8.0 + x*x*x*x*x*90.0;
	return d;
}

__host__ __device__ float newton(complex x0, float eps, int maxiter) {
  complex x = x0;
  int iter = 0;
  while (abs(f(x)) > eps && iter <= maxiter) {
    iter++;
    x = x - f(x)/df(x);
  }  
  return iter;
}


void MathCPU() {
   float xmin = -2, xmax = 2;
   float ymin = -2, ymax = 2;

   int xsteps = width, ysteps = width;
   float 	hx = (xmax - xmin) / xsteps,
				hy = (ymax - ymin) / ysteps; 

   float eps = 0.0001;
   int maxiter = 255;

   float x, y;
   y = ymin;
   for(int i = 0; i < ysteps; i++) {
		x = xmin;
		for(int j = 0; j < xsteps; j++) {
			complex xy = {x,y};
			z[i*width + j] = newton(xy, eps, maxiter);
			x += hx;
		}
		y += hy;
   }		
	
}

__global__ void MathGPUKernel(float *zD) {
	float xi = blockIdx.x * blockDim.x + threadIdx.x;
  	float yi = blockIdx.y * blockDim.y + threadIdx.y;

	float eps = 0.0001;
   int maxiter = 255;

	complex xy = {xi/width*4 - 2,yi/width * 4 - 2};
	zD[(int)yi*width + (int)xi] = newton(xy, eps, maxiter);
}

void MathGPU() {
	const int block_width = 16;
	
	int size = width * width * sizeof(float);
	float *zD; 
	
	hipMalloc(&zD, size); 
	
	dim3 dimGrid(width / block_width, width / block_width); 
	dim3 dimBlock(block_width, block_width); 
	MathGPUKernel <<<dimGrid, dimBlock>>> (zD);
	
	hipMemcpy(z, zD, size, hipMemcpyDeviceToHost);
		
	hipFree(zD);
}

void Display() { 
	glClear(GL_COLOR_BUFFER_BIT);
  	glBegin(GL_POINTS);

   float xmin = -2, xmax = 2;
   float ymin = -2, ymax = 2;

   int xsteps = width, ysteps = width;
   float 	hx = (xmax - xmin) / xsteps,
				hy = (ymax - ymin) / ysteps;
				
	float x, y;
   
  	float max = z[0];
  	float min = z[0];

  	for(int i = 0; i < width; i++) {
    	for(int j = 0; j < width; j++) {
      	if(z[i * width + j] >= max) max = z[i * width + j];
      	if(z[i * width + j] <= min) min = z[i * width + j];
    	}
	}
				
   y = ymin; 
   for(int i = 0; i < width; i++) {
		x = xmin;
		for(int j = 0; j < width; j++) {
			float color = (z[i*width + j] - min) / (max - min);
  			 	glColor3d(	fmod(color * 13, 1.0f),
								fmod(color * 33, 1.0f),
								fmod(color * 49, 1.0f));
			   glVertex2d(x, y);
			x += hx;
		}
		y += hy;
   }
	
	glEnd();
  	glutSwapBuffers();
}

int main(int argc, char **argv) {
  	glutInit(&argc, argv);
  	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
  	glutInitWindowSize(700, 700);
  	glutInitWindowPosition(0, 0);
  	glutCreateWindow("Newton");
  	glClearColor(1.0, 1.0, 1.0, 1.0);
  	glMatrixMode(GL_PROJECTION);
  	glLoadIdentity();
  	glOrtho(-2, 2, 2, -2, 1, -1);

	struct timeval tv;
	double st, end;
	gettimeofday(&tv, NULL);
	st = tv.tv_sec + tv.tv_usec / 1000000.0;
	
	//MathCPU();
	MathGPU();
	
	gettimeofday(&tv, NULL);
	end = tv.tv_sec + tv.tv_usec / 1000000.0;
	printf("Execution time: %.5f с\n", end - st);
	
  	glutDisplayFunc(Display);
  	glutMainLoop();
}